
#include <hip/hip_runtime.h>
#include <stdio.h>

#define RADIUS        3
#define BLOCK_SIZE    256
#define NUM_ELEMENTS  (4096*2)

// CUDA API error checking macro
#define cudaCheck(error) \
  if (error != hipSuccess) { \
    printf("Fatal error: %s at %s:%d\n", \
      hipGetErrorString(error), \
      __FILE__, __LINE__); \
    exit(1); \
  }

__global__ void stencil_1d(int *in, int *out) 
{
    __shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
    int gindex = threadIdx.x + (blockIdx.x * blockDim.x) + RADIUS;
    int lindex = threadIdx.x + RADIUS;

    // Read input elements into shared memory
    temp[lindex] = in[gindex];
    if (threadIdx.x < RADIUS) 
    {
        temp[lindex - RADIUS] = in[gindex - RADIUS];
        temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
    }

    // Make sure all threads get to this point before proceeding!
    __syncthreads();

    // Apply the stencil
    int result = 0;
    for (int offset = -RADIUS ; offset <= RADIUS ; offset++)
        result += temp[lindex + offset];

    // Store the result
    out[gindex-RADIUS] = result;
}

__global__ void stencil_1dV2(int *in, int *out) 
{
    __shared__ int temp[BLOCK_SIZE];
    int gindex = threadIdx.x + (blockIdx.x * blockDim.x) + RADIUS;
    int lindex = threadIdx.x + RADIUS;

    // Read input elements into shared memory
    temp[lindex] = in[gindex];
    if (threadIdx.x < RADIUS) 
    {
        temp[lindex - RADIUS] = in[gindex - RADIUS];
        temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
    }

    // Make sure all threads get to this point before proceeding!
    __syncthreads();

    // Apply the stencil
    int result = 0;
    for (int offset = -RADIUS ; offset <= RADIUS ; offset++)
        result += temp[lindex + offset];

    // Store the result
    out[gindex-RADIUS] = result;
}
int main()
{
  unsigned int i;
  int h_in[NUM_ELEMENTS + 2 * RADIUS], h_out[NUM_ELEMENTS];
  int *d_in, *d_out;

  // Initialize host data
  for( i = 0; i < (NUM_ELEMENTS + 2*RADIUS); ++i )
    h_in[i] = 1; // With a value of 1 and RADIUS of 3, all output values should be 7

  // Allocate space on the device
  cudaCheck( hipMalloc( &d_in, (NUM_ELEMENTS + 2*RADIUS) * sizeof(int)) );
  cudaCheck( hipMalloc( &d_out, NUM_ELEMENTS * sizeof(int)) );

  // Copy input data to device
  cudaCheck( hipMemcpy( d_in, h_in, (NUM_ELEMENTS + 2*RADIUS) * sizeof(int), hipMemcpyHostToDevice) );
  
  hipEvent_t startEvent, stopEvent;
  cudaCheck( hipEventCreate(&startEvent) );
  cudaCheck( hipEventCreate(&stopEvent) );
  float ms;
  cudaCheck( hipEventRecord(startEvent, 0) );

  stencil_1d<<< (NUM_ELEMENTS + BLOCK_SIZE - 1)/BLOCK_SIZE, BLOCK_SIZE >>> (d_in, d_out);
  
  cudaCheck( hipEventRecord(stopEvent, 0) );
  cudaCheck( hipEventSynchronize(stopEvent) );
  cudaCheck( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  fprintf(stderr,"\n----------------------------------\n");
  fprintf(stderr,"add<<<1,1>>>  GPU time is taken=%f ms\n",ms);
  
  cudaCheck( hipEventRecord(startEvent, 0) );

  stencil_1d<<< (NUM_ELEMENTS + BLOCK_SIZE - 1)/BLOCK_SIZE, BLOCK_SIZE >>> (d_in, d_out);
  
  cudaCheck( hipEventRecord(stopEvent, 0) );
  cudaCheck( hipEventSynchronize(stopEvent) );
  cudaCheck( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  fprintf(stderr,"\n----------------------------------\n");
  fprintf(stderr,"add<<<1,1>>>  GPU V2 time is taken=%f ms\n",ms);
  
  cudaCheck( hipMemcpy( h_out, d_out, NUM_ELEMENTS * sizeof(int), hipMemcpyDeviceToHost) );

  // Verify every out value is 7
  for( i = 0; i < NUM_ELEMENTS; ++i )
    if (h_out[i] != 7)
    {
      printf("Element h_out[%d] == %d != 7\n", i, h_out[i]);
      break;
    }

  if (i == NUM_ELEMENTS)
    printf("SUCCESS!\n");

  // Free out memory
  hipFree(d_in);
  hipFree(d_out);

  return 0;
}

