
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// const int TILE_DIM = 32;
// Kernel function to add the elements of two arrays
__global__ void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}

__global__ void add1(int n, float *x, float *y)
{
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int i = index; i < n; i += stride)
      y[i] = x[i] + y[i];
}
__global__ void add2(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}


inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}
int main(void)
{
  int N = 1<<20;  //1M
  float *x, *y;
  
  // Set execution configuration parameters
	//		thr_per_blk: number of CUDA threads per grid block
	//		blk_in_grid: number of blocks in grid
	int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;  
                    //(N-1)/blockSize+1=(((2^20)-1)/2^8)+1=2^12+1=4096
  
  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }
  hipEvent_t startEvent, stopEvent;
  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );
  float ms;
  checkCuda( hipEventRecord(startEvent, 0) );

  // Run kernel on 1M elements on the GPU
  add<<<1, 1>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  // cudaDeviceSynchronize();
  
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  fprintf(stderr,"\n----------------------------------\n");
  fprintf(stderr,"add<<<1,1>>>  GPU time is taken=%f ms\n",ms);
  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;
  
  checkCuda( hipEventRecord(startEvent, 0) );

  add1<<<1, blockSize>>>(N, x, y);    //blockSize=256
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  fprintf(stderr,"\n----------------------------------\n");
  fprintf(stderr,"add1<<<1,256>>>  GPU time is taken=%f ms\n",ms);
  // Check for errors (all values should be 3.0f)
  maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;
  
  checkCuda( hipEventRecord(startEvent, 0) );

  add2<<<numBlocks, blockSize>>>(N, x, y);    //blockSize=256 , numBlocks=4096
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  fprintf(stderr,"\n----------------------------------\n");
  fprintf(stderr,"add2<<<4096,256>>>  GPU time is taken=%f ms\n",ms);
  // Check for errors (all values should be 3.0f)
  maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;
  
  
  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}