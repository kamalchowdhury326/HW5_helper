#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <unistd.h>

#include "cuda_error.h"
#include "default_values.h"
#include "common.h"
#include "cuda_common.h"


//Not pretty, but cc 1.1 doesn't allow to compile kernels in different files, and cc 2.0 does not allow to force noinline. 
#include "cuda_common.cu"


__global__ void moveKernel(plattice1D d_lattice, plattice1D d_lattice_new, int size_i, int size_j, int neighs, int halo){
	int count=0;
	//Los desplazamientos son por los vecinos ghost
	int col = blockDim.x * blockIdx.x + threadIdx.x + neighs;
	int row = blockDim.y * blockIdx.y + threadIdx.y + neighs;
	int my_id = row * (size_i+halo) + col; 

	if (col < size_i+neighs && row < size_j+neighs){
		count = count_neighs(my_id, size_i, d_lattice, neighs, halo);
		check_rules(my_id, count, d_lattice, d_lattice_new);
	}
}


int main(int argc, char **argv){
	int iter=0;

	timing recorded_time = {.fill = -1, .step_init=-1, .step_end = 0, .comm_init = -1, .comm_end = 0, 
							.evolve = -1, .output_init = -1, .output_end = 0, .total = 0, };
    record_time(10, &recorded_time);

	parameters vars;
	read_input_parameters(argc, argv, &vars);
 	
 	hipDeviceSetCacheConfig(hipFuncCachePreferL1);
	
	size_t size = vars.sizesqr * sizeof(TYPE);

	plattice1D lattice = (plattice1D) malloc(size);
	
	RECORD_TIME(0, &recorded_time);
	fill_lattice(lattice, &vars);
	RECORD_TIME(0, &recorded_time);

	plattice1D d_lattice_new;
	plattice1D d_lattice_tmp;
	plattice1D d_lattice;
	CudaSafeCall(hipMalloc(&d_lattice, size));
	CudaSafeCall(hipMalloc(&d_lattice_new, size));
	CudaSafeCall(hipMemcpy(d_lattice, lattice, size, hipMemcpyHostToDevice));


	dim3 threadsPerBlock = select_threadsPerBlock(vars.blocksize);
	//dim3 numBlocks(((vars.size_i) / threadsPerBlock.x), ((vars.size_j) / threadsPerBlock.y));
 	dim3 numBlocks(ceil(((float)vars.size_i) / (float)threadsPerBlock.x), ceil((float)vars.size_j / (float)threadsPerBlock.y));

    dim3 ghostBlockSize(vars.blocksize);
    dim3 ghostRowsGridSize((int)ceil(vars.size_i/(float)ghostBlockSize.x));
    dim3 ghostColsGridSize((int)ceil((vars.size_j+vars.halo)/(float)ghostBlockSize.x));


	RECORD_TIME(2, &recorded_time);
	while (iter <= vars.max_iter){
		if (iter % vars.output_steps == 0) {
			RECORD_TIME(4, &recorded_time);
			write_output(iter, lattice, d_lattice, &vars);
			RECORD_TIME(5, &recorded_time);
		}

		RECORD_TIME(6,&recorded_time);
		copy_Rows<<<ghostRowsGridSize, ghostBlockSize>>>(vars.size_i, d_lattice, vars.neighs, vars.halo);
    	copy_Cols<<<ghostColsGridSize, ghostBlockSize>>>(vars.size_i, d_lattice, vars.neighs, vars.halo);
    	RECORD_TIME(7,&recorded_time);

		//write_output(iter, lattice, d_lattice, &vars);


    	RECORD_TIME(8,&recorded_time);
		moveKernel<<<numBlocks, threadsPerBlock>>>(d_lattice, d_lattice_new, vars.size_i, vars.size_j, vars.neighs, vars.halo);
		hipDeviceSynchronize();
		CudaCheckError();

		d_lattice_tmp = &d_lattice[0];
		d_lattice = &d_lattice_new[0];
		d_lattice_new = &d_lattice_tmp[0];
		
		RECORD_TIME(9,&recorded_time);

		iter++;
	}
	RECORD_TIME(2, &recorded_time);

	hipDeviceReset();
	free(lattice);

	record_time(10, &recorded_time);
	
	output_information(&vars, &recorded_time);
	return 0;
}