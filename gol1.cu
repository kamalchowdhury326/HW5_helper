#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <unistd.h>

#include "cuda_error.h"
#include "default_values.h"
#include "common.h"
#include "cuda_common.h"

//Not pretty, but cc 1.1 doesn't allow to compile kernels in different files, and cc 2.0 does not allow to force noinline. 
#include "cuda_common.cu"


//Los desplazamientos son por los vecinos ghost
/*int iy = blockDim.y * blockIdx.y + threadIdx.y + 1;
int ix = blockDim.x * blockIdx.x + threadIdx.x + 1;
int id = iy * (dim+2) + ix;*/

//Estos define estan porque cuando se usa el data type double se utilizan mas registros y no se puede
//ejecutar la simulacion con bs 1024.
#define my_id (row * (size_i+halo) + col)
#define my_sh_id (threadIdx.x * blockDim.y + threadIdx.y)

__global__ void moveKernel(plattice1D d_lattice, plattice1D d_lattice_new, int size_i, int size_j, int neighs, int halo){
	int count=0;
	int col = (blockDim.x - halo) * blockIdx.x + threadIdx.x;
	int row = (blockDim.y - halo) * blockIdx.y + threadIdx.y; 	
		
	int sh_size_x = blockDim.y;
	
    extern __shared__ TYPE sh_lattice[];

 	if (col < size_i+halo && row < size_j+halo) {
        sh_lattice[my_sh_id] = d_lattice[my_id];
 	}
    __syncthreads();

    // CHECK IF
	/*if (col < size_i+neighs && row < size_j+neighs && 
		threadIdx.x >= (neighs-1) && threadIdx.x < blockDim.x-neighs && 
		threadIdx.y >= (neighs-1) && threadIdx.y < blockDim.y-neighs) {*/
    
    if (col < size_i+neighs && row < size_j+neighs && 
		threadIdx.x >= neighs && threadIdx.x < blockDim.x-neighs && 
		threadIdx.y >= neighs && threadIdx.y < blockDim.y-neighs) {    
        
        count = count_neighs(my_sh_id, sh_size_x-halo, sh_lattice, neighs, halo);	// decrease sh_size_x by 2 to use the same count_neighs function than the rest of the implementations
        check_rules(my_id, count, d_lattice, d_lattice_new);
 	}
}

int main(int argc, char **argv){
	int iter=0;

	timing recorded_time = {.fill = -1, .step_init=-1, .step_end = 0, .comm_init = -1, .comm_end = 0, 
							.evolve = -1, .output_init = -1, .output_end = 0, .total = 0, };
    record_time(10, &recorded_time);

	parameters vars;
	read_input_parameters(argc, argv, &vars);

	size_t size = vars.sizesqr * sizeof(TYPE);

	plattice1D lattice = (plattice1D) malloc(size);
	
	RECORD_TIME(0, &recorded_time);
	fill_lattice(lattice, &vars);
	RECORD_TIME(0, &recorded_time);

	plattice1D d_lattice_new;
	plattice1D d_lattice_tmp;
	plattice1D d_lattice;
	CudaSafeCall(hipMalloc(&d_lattice, size));
	CudaSafeCall(hipMalloc(&d_lattice_new, size));
	CudaSafeCall(hipMemcpy(d_lattice, lattice, size, hipMemcpyHostToDevice));


	dim3 threadsPerBlock = select_threadsPerBlock(vars.blocksize);
	dim3 numBlocks(ceil((float)vars.size_i / (float)(threadsPerBlock.x-vars.halo)), ceil((float)vars.size_j / (float)(threadsPerBlock.y-vars.halo)));
//	dim3 numBlocks((ceil((float)vars.size_i) / (float)threadsPerBlock.x), ceil((float)vars.size_j / (float)threadsPerBlock.y));

	//dim3 threadsPerBlock(BLOCKSIZE_x, BLOCKSIZE_y, 1);
	//dim3 numBlocks(ceil(vars.size_i / (float)(BLOCKSIZE_x-2)), ceil(vars.size_j / (float)(BLOCKSIZE_y-2)));

	size_t sharedsize = sizeof(TYPE) * (threadsPerBlock.x * threadsPerBlock.y);

	printf("sharedsize=%d\n", sharedsize );

    dim3 ghostBlockSize(vars.blocksize);
    dim3 ghostRowsGridSize((int)ceil(vars.size_i/(float)ghostBlockSize.x));
    dim3 ghostColsGridSize((int)ceil((vars.size_i+vars.halo)/(float)ghostBlockSize.x));


	RECORD_TIME(2, &recorded_time);
	while (iter <= vars.max_iter){
		if (iter % vars.output_steps == 0) {
			RECORD_TIME(4, &recorded_time);
			write_output(iter, lattice, d_lattice, &vars);
			RECORD_TIME(5, &recorded_time);
		}


		RECORD_TIME(6,&recorded_time);
		copy_Rows<<<ghostRowsGridSize, ghostBlockSize>>>(vars.size_i, d_lattice, vars.neighs, vars.halo);
    	copy_Cols<<<ghostColsGridSize, ghostBlockSize>>>(vars.size_i, d_lattice, vars.neighs, vars.halo);
    	RECORD_TIME(7,&recorded_time);
    	
    	RECORD_TIME(8,&recorded_time);
		moveKernel<<<numBlocks, threadsPerBlock, sharedsize>>>(d_lattice, d_lattice_new, vars.size_i, vars.size_j, vars.neighs, vars.halo);
		hipDeviceSynchronize();
		CudaCheckError();

		d_lattice_tmp = &d_lattice[0];
		d_lattice = &d_lattice_new[0];
		d_lattice_new = &d_lattice_tmp[0];
		
		RECORD_TIME(9,&recorded_time);

		iter++;
	}
	RECORD_TIME(2, &recorded_time);

	hipDeviceReset();
	free(lattice);

	record_time(10, &recorded_time);
	
	output_information(&vars, &recorded_time);
	return 0;
}