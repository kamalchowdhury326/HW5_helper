#include <stdio.h>
#include<hip/hip_runtime.h>

__global__ void helloCUDA()
{
    printf("Hello, CUDA!\n");
}

int main()
{
    helloCUDA<<<2, 8>>>();
    hipDeviceSynchronize();
    return 0;
}